#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <fstream>
#include <cstdlib>
#include <cmath>

using namespace std;

#define CSC(call)  													\
do {																\
	hipError_t res = call;											\
	if (res != hipSuccess) {										\
		fprintf(stderr, "ERROR in %s:%d. Message: %s\n",			\
				__FILE__, __LINE__, hipGetErrorString(res));		\
		exit(0);													\
	}																\
} while(0)

texture<uchar4, 2, hipReadModeElementType> tex;

__global__	void	kernel(uchar4* out, int w, int h)
{
	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	int idy = blockDim.y * blockIdx.y + threadIdx.y;
	int offsetx = blockDim.x * gridDim.x;
	int offsety = blockDim.y * gridDim.y;

	for (int y = idy; y < h; y += offsety)
		for (int x = idx; x < w; x += offsetx)
		{
			double Gx = 0.0;
			double Gy = 0.0;
			uchar4 pixel;

			pixel = tex2D(tex, x - 1, y - 1);
			Gx -= (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x - 1, y);
			Gx -= (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x - 1, y + 1);
			Gx -= (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x + 1, y - 1);
			Gx += (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x + 1, y);
			Gx += (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x + 1, y + 1);
			Gx += (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);

			pixel = tex2D(tex, x - 1, y - 1);
			Gy -= (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x, y - 1);
			Gy -= (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x + 1, y - 1);
			Gy -= (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x - 1, y + 1);
			Gy += (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x, y + 1);
			Gy += (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);
			pixel = tex2D(tex, x + 1, y + 1);
			Gy += (0.299 * pixel.x) + (0.587 * pixel.y) + (0.114 * pixel.z);

			unsigned char grad = (unsigned char)min((int)sqrt(Gx * Gx + Gy * Gy), (int)0xFF);

			out[y * w + x] = make_uchar4(grad, grad, grad, 0);
		}
}

int		main(void)
{
	int		w, h;
	char path_in[256];
	char path_out[256];

	scanf("%s", path_in);
	scanf("%s", path_out);
	FILE *fp = fopen(path_in, "rb");
	fread(&w, sizeof(int), 1, fp);
	fread(&h, sizeof(int), 1, fp);
	uchar4 *data = (uchar4 *)malloc(sizeof(uchar4) * w * h);
	fread(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	hipArray	*arr;
	hipChannelFormatDesc ch = hipCreateChannelDesc<uchar4>();
	CSC(hipMallocArray(&arr, &ch, w, h));
	CSC(hipMemcpyToArray(arr, 0, 0, data, sizeof(uchar4) * w * h, hipMemcpyHostToDevice));

	tex.addressMode[0] = hipAddressModeClamp;
	tex.addressMode[1] = hipAddressModeClamp;
	tex.channelDesc = ch;
	tex.filterMode = hipFilterModePoint;
	tex.normalized = false;

	CSC(hipBindTextureToArray(tex, arr, ch));

	uchar4 *dev_out;
	CSC(hipMalloc(&dev_out, sizeof(uchar4) * w * h));

	kernel<<<dim3(16, 16), dim3(16, 16)>>>(dev_out, w, h);
	CSC(hipGetLastError());

	CSC(hipMemcpy(data, dev_out, sizeof(uchar4) * w * h, hipMemcpyDeviceToHost));
	CSC(hipUnbindTexture(tex));

	CSC(hipFreeArray(arr));
	CSC(hipFree(dev_out));

	fp = fopen(path_out, "wb");
	fwrite(&w, sizeof(int), 1, fp);
	fwrite(&h, sizeof(int), 1, fp);
	fwrite(data, sizeof(uchar4), w * h, fp);
	fclose(fp);

	free(data);
	return 0;
}
