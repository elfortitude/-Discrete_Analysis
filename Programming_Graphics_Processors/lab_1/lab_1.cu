
#include <hip/hip_runtime.h>
#include <iostream>
#include <cmath>

using namespace std;

__global__ void		kernel(double *arr, int n)
{
	int index, offset;
	index = blockDim.x * blockIdx.x + threadIdx.x;
	offset = blockDim.x * gridDim.x;
	for (int i = index; i < n; i += offset)
		arr[i] = abs(arr[i]);
}

int		main(void)
{
	int		n;
	double  *arr;

	cin >> n;
	arr = (double *)malloc(n * sizeof(double));
	for (int i = 0; i < n; ++i)
		cin >> arr[i];

	double	*dev_arr;
	hipMalloc(&dev_arr, sizeof(double) * n);
	hipMemcpy(dev_arr, arr, sizeof(double) * n, hipMemcpyHostToDevice);

	kernel<<<256, 256>>>(dev_arr, n);

	hipMemcpy(arr, dev_arr, sizeof(double) * n, hipMemcpyDeviceToHost);
	hipFree(dev_arr);
	for (int i = 0; i < n; ++i)
		printf("%.10e ", arr[i]);
	printf("\n");
	free(arr);
	return (0);
}
